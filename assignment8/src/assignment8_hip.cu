#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cerrno>
#include <chrono>
#include <algorithm>
#include <omp.h>
#include <hip/hip_runtime.h>
#include "cudautil_hip.h"
/*
*TODO: find the best BLOCK_SIZE 
*/
#define BLOCK_SIZE 8
/*
*TODO: find the best TILEDIM 
*/
#define TILE_DIM 8

/*
*TODO: Task b: Global memory MM implementation 
*/
__global__ void MM(double* a, double* b, double* c,
		int N, int REP ) {

    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    double sum = 0.0;
    for(int r = 0 ; r< REP; ++r)
        if( col < N && row < N) 
        {
            for(int i = 0; i < N; i++) 
            {
                    sum += a[row * N + i] * b[i * N + col];
            }
            c[row * N + col] = sum;
        }
}

/*
*TODO: Task d: Shared memory MM implementation 
*/
__global__ void sharedTiledMM(double* a, double* b, double* c,
		int N, int REP) {
	__shared__ double aTile[TILE_DIM][TILE_DIM];
	__shared__ double bTile[TILE_DIM][TILE_DIM];
	// double aTile[TILE_DIM][TILE_DIM];
	// double bTile[TILE_DIM][TILE_DIM];
	int row = blockIdx.y* blockDim.y+ threadIdx.y; 
	int col= blockIdx.x* blockDim.x+ threadIdx.x; 
	double sum = 0; 
	
	for(int j = 0 ; j<REP ; ++j){
		for (int k = 0; k < N; k += TILE_DIM) { 
			aTile[threadIdx.y][threadIdx.x] = a[ (row * N) + k + threadIdx.x]; 
			bTile[threadIdx.y][threadIdx.x] = b[(threadIdx.y + k)*N + col]; 
			__syncthreads(); 
			for (int i = 0; i < TILE_DIM; i++) 
				sum += aTile[threadIdx.y][i]* bTile[i][threadIdx.x];
			__syncthreads();
		} 
		c[row*N +col] = sum; 
	}
}


int main(int argc, char *argv[]) {

    /*
       +TODO: Task a: print device properties 
    */
    int device = 0;
    hipSetDevice(device);
    // PrintDeviceInfo();
    
    if (argc < 2) {
        printf("For C(NxN) = A(NxN)* B(NxN), Matrix size value N must be provided ! \n");
        exit(1);
    }

    char *pEnd;
    int N = strtol(argv[1], &pEnd, 10);
    if (errno == ERANGE) {
        printf("Problem with the first number  N .");
        exit(2);
    }
    int REP = 10;
    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);  

    /*Memory allocations and Initializations of maticies*/
    // double *a=(double*)malloc(sizeof(double)*N*N);
    // double *b=(double*)malloc(sizeof(double)*N*N);
    // double *c=(double*)malloc(sizeof(double)*N*N);;
    double *a, *b, *c;
    hipHostMalloc((void**)&a, sizeof(double)*N*N);
    hipHostMalloc((void**)&b, sizeof(double)*N*N);
    hipHostMalloc((void**)&c, sizeof(double)*N*N);

    double *d_a, *d_b, *d_c;
    /*
    * TODO:Task e: Use UVA for device memory  
    */ 
    hipMalloc(&d_a, sizeof(double)*N*N);
    hipMalloc(&d_b, sizeof(double)*N*N);
    hipMalloc(&d_c, sizeof(double)*N*N);

    // Initialization on CPU
#pragma omp parallel for collapse(2) schedule(static)
    for (int i = 0; i < N; ++i) 
        for (int j = 0; j < N; ++j) {
            a[i*N+j] = atan(i*N+j);
            b[i*N+j] = cos(i*N+j);
            c[i*N+j] = 0.0;
    }

    // Copy initial values to GPUs
    auto t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy( d_a, a, sizeof(double)*N*N, hipMemcpyHostToDevice );
    hipMemcpy( d_b, b, sizeof(double)*N*N, hipMemcpyHostToDevice );
    hipMemcpy( d_c, c, sizeof(double)*N*N, hipMemcpyHostToDevice );
    

    using dsec = std::chrono::duration<double>;
    double mf = 2.0*(double)N*(double)N*(double)N*(double)REP*1.0e-6;
    
    // Compute Checksum for Simple Correctness Checks
    // double checksum = cpu_matrix_mult_checksum(a, b, N, REP);

    /*
    * Basic MM Kernel Call & Time Measurements
    */
    dim3 dimBlockMM(N,N);


    // auto t0 = std::chrono::high_resolution_clock::now();
    // MM <<<dimGrid, dimBlock >>>(d_a, d_b ,d_c, N, REP);
    // hipDeviceSynchronize();
    // auto t1 = std::chrono::high_resolution_clock::now();

    // //Calculate Flops/sec,
    // double dur = std::chrono::duration_cast<dsec>(t1-t0).count();
    // std::cout<<"MM MFlops/s(N*N="<< N*N <<" ): "<<mf/dur<<std::endl;
    // // // Copy the result back to CPU & correctness check
    // hipMemcpy( c, d_c, sizeof(double)*N*N, hipMemcpyDeviceToHost );
    // // Checksum ( N, c, checksum );

    // //reset_result_array d_c
    // #pragma omp parallel for collapse(2) schedule(static)
    // for (int i = 0; i < N; ++i) 
    //     for (int j = 0; j < N; ++j) {
    //         c[i*N+j] = 0.0;
    // }
    // hipMemcpy( d_c, c, sizeof(double)*N*N, hipMemcpyHostToDevice );
    
    /*    
    *Basic Tiled MM with Shared Memory Kernel Call & Time Measurements
    */
    // auto t2 = std::chrono::high_resolution_clock::now();
    sharedTiledMM <<<dimGrid, dimBlock >>>(d_a, d_b ,d_c, N, REP);
    hipDeviceSynchronize();
    // auto t3 = std::chrono::high_resolution_clock::now();

    //double dur = std::chrono::duration_cast<dsec>(t3-t2).count();
    //Calculate Flops/sec, Correctness Checks & Reset Result array C
    // Copy the result back to CPU & correctness check
    hipMemcpy( c, d_c, sizeof(double)*N*N, hipMemcpyDeviceToHost );
    auto t3 = std::chrono::high_resolution_clock::now();
    double dur = std::chrono::duration_cast<dsec>(t3-t2).count();
    std::cout<<"Shared Tiled MFlops/s for (N*N="<< N*N <<"): "<<mf/dur<<std::endl;
    // Checksum ( N, c, checksum );
    hipFree(a);
    hipFree(b);
    hipHostFree(c);
    // free(a);
    // free(b);
    // free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}

